
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

extern "C" {
    void executeFFT(float* d_input, hipfftComplex* d_output, int signalSize) {
        hipfftHandle plan;
        hipfftPlan1d(&plan, signalSize, HIPFFT_R2C, 1);
        hipfftExecR2C(plan, d_input, d_output);
        hipfftDestroy(plan);
    }
}
